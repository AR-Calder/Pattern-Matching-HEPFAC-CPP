#include "hip/hip_runtime.h"
#include <iostream>
// TODO try with pitched, 2d texture etc.
/*
PITCHED
https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1g32bd7a39135594788a542ae72217775c

TEXTURE (GENERAL)
http://www.drdobbs.com/parallel/cuda-supercomputing-for-the-masses-part/218100902?pgno=2
https://stackoverflow.com/questions/13119813/bound-cuda-texture-reads-zero/13120722#13120722

http://www.subdude-site.com/WebPages_Local/RefInfo/Computer/Linux/LinuxGuidesOfOthers/linuxProgrammingGuides/pdfs/3Dgpu/3D_GPGPU_beginners_tutorial_2009_155pgs.pdf

*/

texture<int, 1, hipReadModeElementType> tex_ref;
hipArray* cuda_array;

__global__ void test(int* results){
    const int tidx = threadIdx.x;
    results[tidx] = tex1D(tex_ref, tidx) * 3;
}

int main(){
    int *host_arr;
    int host_arr_size = 8;

    // Create and populate host array
    hipHostMalloc((void**)&host_arr, host_arr_size * sizeof(int));
    for (int i = 0; i < host_arr_size; ++i){
        host_arr[i] = i * 2;
        std::cout << host_arr[i] << std::endl;
    }

    // bind to texture
    hipChannelFormatDesc cuDesc = hipCreateChannelDesc <int >();
    hipMallocArray(&cuda_array, &cuDesc, host_arr_size);
    hipMemcpyToArray(cuda_array, 0, 0, host_arr , host_arr_size * sizeof(int), hipMemcpyHostToDevice);
    hipBindTextureToArray(tex_ref , cuda_array);
    // Allocate results array
    int * result_arr;
    hipMalloc((void**)&result_arr, host_arr_size*sizeof(int));

    // launch kernel
    test<<<1, host_arr_size>>>(result_arr);

    // fetch results
    hipMemcpy(host_arr, result_arr, host_arr_size * sizeof(int), hipMemcpyDeviceToHost);
    // print results
    for (int i = 0; i < host_arr_size; ++i){
        std::cout << host_arr[i] << std::endl;
    }

    // Tidy Up
    hipUnbindTexture(tex_ref);
    hipHostFree(host_arr);
    hipFreeArray(cuda_array);
    hipFree(result_arr);
}
